#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "common.h"

#define NUM_THREADS 128
#define InitialCapacity 10
#define binSize 0.01

extern double size;
//
//  benchmarking program
//

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}

__device__ int indexing(int bin_ind_m, int bin_ind_n, int lda, int local_ind){
    return (bin_ind_m+lda*bin_ind_n)*InitialCapacity+local_ind;
}

__global__ void move_gpu (particle_t * particles, int n, double size, int* bin_list, int* bin_num, int lda)
{

  // Get thread (particle) ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;

    particle_t * p = &particles[tid];
    int m_old = (p->x)/binSize, n_old = (p->y)/binSize;
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

    int m_new = (p->x)/binSize, n_new = (p->y)/binSize;
    if(m_new != m_old || n_new != n_old){
        for(int i = 0; i < bin_num[m_old + n_old * lda]; i++){
            if(bin_list[indexing(m_old, n_old, lda, i)] == tid){
                bin_list[indexing(m_old, n_old, lda, i)] = -1;
                break;
            }
        }

        int index = atomicAdd(&bin_num[m_new + n_new * lda], 1);
        bin_list[indexing(m_new, n_new, lda, index)] = tid;
    }
}

__global__ void clean_bins(int* bin_list, int* bin_num, int lda){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= lda * lda) return;

    int ind = 0;
    for(int i = 0; i < bin_num[tid]; i++){
        if(bin_list[tid * InitialCapacity + i] != -1){
            bin_list[tid * InitialCapacity + ind] = bin_list[tid * InitialCapacity + i];
            ind++;
        }
    }
    bin_num[tid] = ind;
}

__global__ void init_bin_num(int* bin_num, int total_bin){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= total_bin) return;
    bin_num[tid] = 0;
}

__global__ void init_bin_list(int* bin_list, int* bin_num, particle_t* particles,
        int n, int lda){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;
    particle_t& temp = particles[tid];
    int m_ind = temp.x/binSize, n_ind = temp.y/binSize;
    int local_ind = atomicAdd(&bin_num[m_ind+n_ind*lda], 1);
    bin_list[indexing(m_ind, n_ind, lda, local_ind)] = tid;
}

__global__ void compute_forces_gpu(int* bin_list, int*bin_num, particle_t* particles,
        int n, int lda){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= n) return;
    particles[tid].ax = particles[tid].ay = 0;
    int m_ind = particles[tid].x/binSize, n_ind = particles[tid].y/binSize;
    for(int m = max(m_ind-1, 0); m <= min(m_ind+1, lda-1); m++){
        for(int n = max(n_ind-1, 0); n <= min(n_ind+1, lda-1); n++){
            for(int i = 0; i < bin_num[m+n*lda]; i++){
                apply_force_gpu(particles[tid], particles[bin_list[indexing(m, n, lda, i)]]);
            }
        }
    }
}

int main( int argc, char **argv )
{
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize();

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }

    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );

    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    set_size( n );

    int lda_bin = ceil(size/binSize);
    int total_bin = lda_bin*lda_bin;

    init_particles( n, particles );

    int* bin_list; hipMalloc((void **) &bin_list, total_bin*InitialCapacity*sizeof(int));
    int* bin_num; hipMalloc((void **) &bin_num, total_bin*sizeof(int));
    int particle_blks = (n + NUM_THREADS - 1) / NUM_THREADS;
    int bin_blks = (total_bin + NUM_THREADS - 1) / NUM_THREADS;

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;

    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    init_bin_num <<< bin_blks, NUM_THREADS >>> (bin_num, total_bin);
    init_bin_list <<< particle_blks, NUM_THREADS >>> (bin_list, bin_num, d_particles, n, lda_bin);
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
        //
        //  compute forces
        //
	compute_forces_gpu <<< particle_blks, NUM_THREADS >>> (bin_list, bin_num, d_particles, n, lda_bin);

        //
        //  move particles
        //
	move_gpu <<< particle_blks, NUM_THREADS >>> (d_particles, n, size, bin_list, bin_num, lda_bin);
    clean_bins <<< bin_blks, NUM_THREADS >>> (bin_list, bin_num, lda_bin);


    /*init_bin_num <<< bin_blks, NUM_THREADS >>> (bin_num, bin_add_num, total_bin);*/
    /*init_bin_list <<< particle_blks, NUM_THREADS >>> (bin_list, bin_num, d_particles, n, lda_bin);*/
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
	    // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
	}
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;

    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );

    free( particles );
    hipFree(d_particles);
    hipFree(bin_list); hipFree(bin_num);
    if( fsave )
        fclose( fsave );

    return 0;
}
