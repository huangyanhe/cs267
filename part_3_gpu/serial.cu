#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include "common.h"

#define InitialCapacity 10
#define RegionSize 0.01
extern double size;
int indexing(int region_ind_m, int region_ind_n, int lda, int local_ind){
    return (region_ind_m+lda*region_ind_n)*InitialCapacity+local_ind;
}

void init_region_num(int* region_num, int total_region){
    for(int tid = 0; tid < total_region; tid++)
    region_num[tid] = 0;
}

void init_region_list(int* region_list, int* region_num, particle_t* particles, int n, int lda){
    for(int tid = 0; tid < n; tid++){
        particle_t& temp = particles[tid];
        int m_ind = temp.x/RegionSize, n_ind = temp.y/RegionSize;
        int local_ind = region_num[m_ind+n_ind*lda];
        region_num[m_ind+n_ind*lda]++;
        region_list[indexing(m_ind, n_ind, lda, local_ind)] = tid;
    }
}

void compute_forces(int* region_list, int*region_num, particle_t* particles, int n, int lda){
    for(int tid = 0; tid < n; tid++){
        particles[tid].ax = particles[tid].ay = 0;
        int m_ind = particles[tid].x/RegionSize, n_ind = particles[tid].y/RegionSize;
        for(int m = max(m_ind-1, 0); m <= min(m_ind+1, lda-1); m++){
            for(int n = max(n_ind-1, 0); n <= min(n_ind+1, lda-1); n++){
                for(int i = 0; i < region_num[m+n*lda]; i++){
                    apply_force(particles[tid], particles[region_list[indexing(m, n, lda, i)]]);
                }
            }
        }
    }
}

//
//  benchmarking program
//
int main( int argc, char **argv )
{    
    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );
    set_size( n );

    int lda_region = ceil(size/RegionSize);
    int total_region = lda_region*lda_region;

    init_particles( n, particles );

    int* region_list = (int*) malloc(total_region*InitialCapacity*sizeof(int)); 
    int* region_num = (int*) malloc(total_region*sizeof(int));
    
    //
    //  simulate a number of time steps
    //
    double simulation_time = read_timer( );
    for( int step = 0; step < NSTEPS; step++ )
    {
        init_region_num(region_num, total_region);
        init_region_list(region_list, region_num, particles, n, lda_region);
        //
        //  compute forces
        //
        compute_forces(region_list, region_num, particles, n, lda_region);
        
        //
        //  move particles
        //
        for( int i = 0; i < n; i++ ) 
            move( particles[i] );
        
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 )
            save( fsave, n, particles );
    }
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    free(region_list); free(region_num);
    if( fsave )
        fclose( fsave );
    
    return 0;
}
